#include "../cuda_array_mapper.h"
#include "../../src/placement/to_gray.h"

unsigned int toGrayUsingCuda(unsigned int value)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  std::vector<unsigned int> data = { value };
  auto arrayProvider =
      std::make_shared<CudaArrayMapper<unsigned int>>(1, 1, data, channelDesc);

  ToGray(arrayProvider).runKernel();

  auto resultVector = arrayProvider->copyDataFromGpu();

  return resultVector[0];
}

#include "../cuda_array_mapper.h"
#include "../../src/placement/to_gray.h"

unsigned int toGrayUsingCuda(unsigned int value)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  std::vector<unsigned int> data = { value };
  auto arrayProvider =
      std::make_shared<CudaArrayMapper<unsigned int>>(1, 1, data, channelDesc);

  toGray(arrayProvider, 1);

  auto resultVector = arrayProvider->copyDataFromGpu();

  return resultVector[0];
}

#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <vector>
#include "../../src/placement/distance_transform.h"
#include "../../src/utils/cuda_helper.h"
#include "../cuda_array_mapper.h"

void callApollonoius(std::vector<Eigen::Vector4f> &image)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipArray_t array;
  int labelCount = 1;
  int imageSize = 4;
  int pixelCount = imageSize * imageSize;
  HANDLE_ERROR(hipMallocArray(&array, &channelDesc, imageSize, imageSize,
                               hipArraySurfaceLoadStore));
  HANDLE_ERROR(hipMemcpyToArray(array, 0, 0, image.data(),
                                 pixelCount * sizeof(Eigen::Vector4f),
                                 hipMemcpyHostToDevice));

  thrust::device_vector<float4> seedBuffer(pixelCount, make_float4(0, 0, 0, 0));
  seedBuffer[0] = make_float4(1, 1, 1, 1);
  thrust::device_vector<float> distanceVector;
  thrust::device_vector<int> computeVector;
  thrust::device_vector<int> computeVectorTemp;
  thrust::device_vector<int> computeSeedIds;
  thrust::device_vector<int> computeSeedIndices;

  cudaJFAApolloniusThrust(array, imageSize, labelCount, seedBuffer,
                          distanceVector, computeVector, computeVectorTemp,
                          computeSeedIds, computeSeedIndices);

  HANDLE_ERROR(hipMemcpyFromArray(image.data(), array, 0, 0,
                                   pixelCount * sizeof(Eigen::Vector4f),
                                   hipMemcpyDeviceToHost));

  hipFree(array);
}

std::vector<Eigen::Vector4f> callDistanceTransform(
    std::shared_ptr<CudaArrayMapper<float>> depthImageProvider,
    std::vector<float> &result)
{
  depthImageProvider->map();

  hipArray_t outputArray;
  hipChannelFormatDesc outputChannelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  HANDLE_ERROR(hipMallocArray(
      &outputArray, &outputChannelDesc, depthImageProvider->getWidth(),
      depthImageProvider->getHeight(), hipArraySurfaceLoadStore));

  thrust::device_vector<int> computeVector;
  thrust::device_vector<float> resultVector;

  cudaJFADistanceTransformThrust(
      depthImageProvider->getArray(), depthImageProvider->getChannelDesc(),
      outputArray, depthImageProvider->getWidth(),
      depthImageProvider->getWidth(), depthImageProvider->getHeight(),
      computeVector, resultVector);

  int pixelCount =
      depthImageProvider->getWidth() * depthImageProvider->getHeight();
  std::vector<Eigen::Vector4f> resultImage(pixelCount);
  HANDLE_ERROR(hipMemcpyFromArray(resultImage.data(), outputArray, 0, 0,
                                   pixelCount * sizeof(Eigen::Vector4f),
                                   hipMemcpyDeviceToHost));

  depthImageProvider->unmap();
  hipFree(outputArray);

  thrust::host_vector<float> resultHost = resultVector;
  for (auto element : resultHost)
    result.push_back(element);

  return resultImage;
}


#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <vector>
#include "../../src/placement/apollonius.h"
#include "../../src/utils/cuda_helper.h"
#include "../cuda_array_mapper.h"

std::vector<int> callApollonoius(std::vector<Eigen::Vector4f> &image,
                                 std::vector<float> distances, int imageSize,
                                 std::vector<Eigen::Vector4f> labelsSeed)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  int labelCount = labelsSeed.size();
  auto imageMapper = std::make_shared<CudaArrayMapper<Eigen::Vector4f>>(
      imageSize, imageSize, image, channelDesc);

  hipChannelFormatDesc channelDescDistances =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  auto distancesMapper = std::make_shared<CudaArrayMapper<float>>(
      imageSize, imageSize, distances, channelDescDistances);

  thrust::device_vector<float4> seedBuffer(labelCount, make_float4(0, 0, 0, 0));
  for (size_t i = 0; i < labelCount; ++i)
    seedBuffer[i] = make_float4(labelsSeed[i].x(), labelsSeed[i].y(),
                                labelsSeed[i].z(), labelsSeed[i].w());

  Apollonius apollonius(distancesMapper, imageMapper, seedBuffer, labelCount);
  apollonius.run();

  image = imageMapper->copyDataFromGpu();

  imageMapper->unmap();

  std::vector<int> result;
  thrust::host_vector<int> labelIndices = apollonius.getIds();
  for (auto index : labelIndices)
    result.push_back(index);

  return result;
}


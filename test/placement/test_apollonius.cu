#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <vector>
#include "../../src/placement/apollonius.h"
#include "../../src/utils/cuda_helper.h"
#include "../cuda_array_mapper.h"

std::vector<int> callApollonoius(std::vector<Eigen::Vector4f> &image,
                     std::vector<float> distances)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  int labelCount = 1;
  int imageSize = 4;
  auto imageMapper = std::make_shared<CudaArrayMapper<Eigen::Vector4f>>(
      imageSize, imageSize, image, channelDesc);

  hipChannelFormatDesc channelDescDistances =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  auto distancesMapper = std::make_shared<CudaArrayMapper<float>>(
      imageSize, imageSize, distances, channelDescDistances);
  thrust::device_vector<float4> seedBuffer(labelCount, make_float4(0, 0, 0, 0));
  seedBuffer[0] = make_float4(0, 2, 1, 1);

  Apollonius apollonius(distancesMapper, imageMapper, seedBuffer, labelCount);
  apollonius.run();

  image = imageMapper->copyDataFromGpu();

  imageMapper->unmap();

  std::vector<int> result;
  thrust::host_vector<int> labelIndices = apollonius.getIds();
  for (auto index : labelIndices)
    result.push_back(index);

  return result;
}


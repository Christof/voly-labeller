#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <vector>
#include "../../src/placement/apollonius.h"
#include "../../src/utils/cuda_helper.h"
#include "../cuda_array_mapper.h"

void callApollonoius(std::vector<Eigen::Vector4f> &image,
                     std::vector<float> distances)
{
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  int labelCount = 1;
  int imageSize = 4;
  int pixelCount = imageSize * imageSize;
  auto imageMapper = std::make_shared<CudaArrayMapper<Eigen::Vector4f>>(
      imageSize, imageSize, image, channelDesc);

  thrust::device_vector<float4> seedBuffer(pixelCount, make_float4(0, 0, 0, 0));
  seedBuffer[0] = make_float4(1, 1, 1, 1);
  thrust::device_vector<float> distanceVector(distances);

  Apollonius apollonius(imageMapper, seedBuffer, distanceVector, labelCount);
  apollonius.run();

  image = imageMapper->copyDataFromGpu();

  imageMapper->unmap();
}


#include "hip/hip_runtime.h"
#include "./integral_costs_calculator.h"
#include "../utils/cuda_helper.h"

__global__ void sumWeightedCosts(hipTextureObject_t occlusion,
                                 float occlusionWeight,
                                 hipSurfaceObject_t output, int width,
                                 int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  float occlusionValue = tex2D<float>(occlusion, x + 0.5f, y + 0.5f);

  float sum = occlusionWeight * occlusionValue;

  surf2Dwrite(sum, output, x * sizeof(float), y);
}

namespace Placement
{

IntegralCostsCalculator::IntegralCostsCalculator(
    std::shared_ptr<CudaArrayProvider> occlusionProvider,
    std::shared_ptr<CudaArrayProvider> outputProvider)
  : occlusionProvider(occlusionProvider), outputProvider(outputProvider)
{
}

IntegralCostsCalculator::~IntegralCostsCalculator()
{
  if (occlusion)
    hipDestroyTextureObject(occlusion);
  if (output)
    hipDestroySurfaceObject(output);
}

void IntegralCostsCalculator::runKernel()
{
  if (!occlusion)
    createSurfaceObjects();

  int outputWidth = outputProvider->getWidth();
  int outputHeight = outputProvider->getHeight();

  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid(divUp(outputWidth, dimBlock.x), divUp(outputHeight, dimBlock.y),
               1);

  float occlusionWeight = 1.0f;

  sumWeightedCosts << <dimGrid, dimBlock>>>
      (occlusion, occlusionWeight, output, outputWidth, outputHeight);

  HANDLE_ERROR(hipDeviceSynchronize());
}

void IntegralCostsCalculator::createSurfaceObjects()
{
  occlusionProvider->map();
  outputProvider->map();

  auto resDesc = occlusionProvider->getResourceDesc();
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&occlusion, &resDesc, &texDesc, NULL);

  auto outputResDesc = outputProvider->getResourceDesc();
  hipCreateSurfaceObject(&output, &outputResDesc);

  occlusionProvider->unmap();
  outputProvider->unmap();
}

}  // namespace Placement

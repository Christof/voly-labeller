#include "hip/hip_runtime.h"
#include "./summed_area_table.h"
#include <thrust/device_vector.h>
#include <iostream>
#include "../utils/cuda_helper.h"

texture<float, 2, hipReadModeElementType> textureReadDepth;

#define WS 32    // Warp size (defines b x b block size where b = WS)
#define HWS 16   // Half Warp Size
#define DW 8     // Default number of warps (computational block height)
#define CHW 7    // Carry-heavy number of warps
                 // (computational block height for some kernels)
#define OW 6     // Optimized number of warps (computational block height for
                 // some kernels)
#define DNB 6    // Default number of blocks per SM (minimum blocks per SM
                 // launch bounds)
#define ONB 5    // Optimized number of blocks per SM (minimum blocks per SM
                 // for some kernels)
#define MTS 192  // Maximum number of threads per block with 8 blocks per SM
#define MBO 8    // Maximum number of blocks per SM using optimize or maximum
                 // warps
#define CHB 7    // Carry-heavy number of blocks per SM using default number of
                 // warps
#define MW 6     // Maximum number of warps per block with 8 blocks per SM (with
                 // all warps computing)
#define SOW 5    // Dual-scheduler optimized number of warps per block (with
                 // 8 blocks per SMand to use the dual scheduler with 1
                 // computing warp)
#define MBH 3    // Maximum number of blocks per SM using half-warp size

/**
 *  @ingroup gpu
 *  @brief Algorithm SAT stage 1
 *
 *  This function computes the algorithm stage S.1 following:
 *
 *  In parallel for all \f$m\f$ and \f$n\f$, compute and store the
 *  \f$P_{m,n}(\bar{Y})\f$ and \f$P^T_{m,n}(\hat{V})\f$.
 *
 *  @note The CUDA kernel functions (as this one) have many
 *  idiosyncrasies and should not be used lightly.
 *
 *  @see [Nehab:2011] cited in alg5() and figure in algSAT()
 *  @param[in] g_in Input image
 *  @param[out] g_ybar All \f$P_{m,n}(\bar{Y})\f$
 *  @param[out] g_vhat All \f$P^T_{m,n}(\hat{V})\f$
 */
__global__ void algSAT_stage1(const int c_width, const int c_height,
                              const float *g_in, float *g_ybar, float *g_vhat)
{
  const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x,
            by = blockIdx.y, col = bx * WS + tx, row0 = by * WS;

  __shared__ float s_block[WS][WS + 1];

  float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[ty][tx];

  g_in += (row0 + ty) * c_width + col;
  g_ybar += by * c_width + col;
  g_vhat += bx * c_height + row0 + tx;

#pragma unroll
  for (int i = 0; i < WS - (WS % SOW); i += SOW)
  {
    **bdata = *g_in;
    bdata += SOW;
    g_in += SOW * c_width;
  }
  if (ty < WS % SOW)
  {
    **bdata = *g_in;
  }

  __syncthreads();

  if (ty == 0)
  {
    {  // calculate ybar -----------------------
      float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[0][tx];

      float prev = **bdata;
      ++bdata;

#pragma unroll
      for (int i = 1; i < WS; ++i, ++bdata)
        **bdata = prev = **bdata + prev;

      *g_ybar = prev;
    }

    {  // calculate vhat -----------------------
      float *bdata = s_block[tx];

      float prev = *bdata;
      ++bdata;

#pragma unroll
      for (int i = 1; i < WS; ++i, ++bdata)
        prev = *bdata + prev;

      *g_vhat = prev;
    }
  }
}

/**
 *  @ingroup gpu
 *  @brief Algorithm SAT stage 2
 *
 *  This function computes the algorithm stage S.2 following:
 *
 *  Sequentially for each \f$m\f$, but in parallel for each \f$n\f$,
 *  compute and store the \f$P_{m,n}(Y)\f$ and using the previously
 *  computed \f$P_{m,n}(\bar{Y})\f$.  Compute and store
 *  \f$s(P_{m,n}(Y))\f$.
 *
 *  @note The CUDA kernel functions (as this one) have many
 *  idiosyncrasies and should not be used lightly.
 *
 *  @see [Nehab:2011] cited in alg5() and figure in algSAT()
 *  @param[in,out] g_ybar All \f$P_{m,n}(\bar{Y})\f$ fixed to \f$P_{m,n}(Y)\f$
 *  @param[out] g_ysum All \f$s(P_{m,n}(Y))\f$
 */
__global__ void algSAT_stage2(const int c_n_size, const int c_m_size,
                              const int c_width, float *g_ybar, float *g_ysum)
{
  const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x,
            col0 = bx * MW + ty, col = col0 * WS + tx;

  if (col >= c_width)
    return;

  g_ybar += col;
  float y = *g_ybar;
  int ln = HWS + tx;

  if (tx == WS - 1)
    g_ysum += col0;

  volatile __shared__ float s_block[MW][HWS + WS + 1];

  if (tx < HWS)
    s_block[ty][tx] = 0.f;
  else
    s_block[ty][ln] = 0.f;

  for (int n = 1; n < c_n_size; ++n)
  {
    // calculate ysum -----------------------
    s_block[ty][ln] = y;

    s_block[ty][ln] += s_block[ty][ln - 1];
    s_block[ty][ln] += s_block[ty][ln - 2];
    s_block[ty][ln] += s_block[ty][ln - 4];
    s_block[ty][ln] += s_block[ty][ln - 8];
    s_block[ty][ln] += s_block[ty][ln - 16];

    if (tx == WS - 1)
    {
      *g_ysum = s_block[ty][ln];
      g_ysum += c_m_size;
    }

    // fix ybar -> y -------------------------
    g_ybar += c_width;
    y = *g_ybar += y;
  }
}

/**
 *  @ingroup gpu
 *  @brief Algorithm SAT stage 3
 *
 *  This function computes the algorithm stage S.3 following:
 *
 *  Sequentially for each \f$n\f$, but in parallel for each \f$m\f$,
 *  compute and store the \f$P^T{m,n}(V)\f$ using the previously
 *  computed \f$P_{m-1,n}(Y)\f$, \f$P^T_{m,n}(\hat{V})\f$ and
 *  \f$s(P_{m,n}(Y))\f$.
 *
 *  @note The CUDA kernel functions (as this one) have many
 *  idiosyncrasies and should not be used lightly.
 *
 *  @see [Nehab:2011] cited in alg5() and figure in algSAT()
 *  @param[in] g_ysum All \f$s(P_{m,n}(Y))\f$
 *  @param[in,out] g_vhat All \f$P^T_{m,n}(\hat{V})\f$ fixed to
 *  \f$P^T_{m,n}(V)\f$
 */
__global__ void algSAT_stage3(const int c_m_size, const int c_height,
                              const float *g_ysum, float *g_vhat)
{
  const int tx = threadIdx.x, ty = threadIdx.y, by = blockIdx.y,
            row0 = by * MW + ty, row = row0 * WS + tx;

  if (row >= c_height)
    return;

  g_vhat += row;
  float y = 0.f, v = 0.f;

  if (row0 > 0)
    g_ysum += (row0 - 1) * c_m_size;

  for (int m = 0; m < c_m_size; ++m)
  {
    // fix vhat -> v -------------------------
    if (row0 > 0)
    {
      y = *g_ysum;
      g_ysum += 1;
    }

    v = *g_vhat += v + y;
    g_vhat += c_height;
  }
}

/**
 *  @ingroup gpu
 *  @brief Algorithm SAT stage 4
 *
 *  This function computes the algorithm stage S.4 following:
 *
 *  In parallel for all \f$m\f$ and \f$n\f$, compute \f$B_{m,n}(Y)\f$
 *  then compute and store \f$B_{m,n}(V)\f$ and using the previously
 *  computed \f$P_{m,n}(Y)\f$ and \f$P^T_{m,n}(V)\f$.
 *
 *  @note The CUDA kernel functions (as this one) have many
 *  idiosyncrasies and should not be used lightly.
 *
 *  @see [Nehab:2011] cited in alg5() and figure in algSAT()
 *  @param[in,out] g_inout The input and output image
 *  @param[in] g_y All \f$P_{m,n}(Y)\f$
 *  @param[in] g_v All \f$P^T_{m,n}(V)\f$
 */
__global__ void algSAT_stage4(const int c_width, const int c_height,
                              float *g_inout, const float *g_y,
                              const float *g_v)
{
  const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x,
            by = blockIdx.y, col = bx * WS + tx, row0 = by * WS;

  __shared__ float s_block[WS][WS + 1];

  float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[ty][tx];

  g_inout += (row0 + ty) * c_width + col;
  if (by > 0)
    g_y += (by - 1) * c_width + col;
  if (bx > 0)
    g_v += (bx - 1) * c_height + row0 + tx;

#pragma unroll
  for (int i = 0; i < WS - (WS % SOW); i += SOW)
  {
    **bdata = *g_inout;
    bdata += SOW;
    g_inout += SOW * c_width;
  }
  if (ty < WS % SOW)
  {
    **bdata = *g_inout;
  }

  __syncthreads();

  if (ty == 0)
  {
    {  // calculate y -----------------------
      float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[0][tx];

      float prev;
      if (by > 0)
        prev = *g_y;
      else
        prev = 0.f;

#pragma unroll
      for (int i = 0; i < WS; ++i, ++bdata)
        **bdata = prev = **bdata + prev;
    }

    {  // calculate x -----------------------
      float *bdata = s_block[tx];

      float prev;
      if (bx > 0)
        prev = *g_v;
      else
        prev = 0.f;

#pragma unroll
      for (int i = 0; i < WS; ++i, ++bdata)
        *bdata = prev = *bdata + prev;
    }
  }

  __syncthreads();

  bdata = (float(*)[WS + 1]) & s_block[ty][tx];

  g_inout -= (WS - (WS % SOW)) * c_width;

#pragma unroll
  for (int i = 0; i < WS - (WS % SOW); i += SOW)
  {
    *g_inout = **bdata;
    bdata += SOW;
    g_inout += SOW * c_width;
  }
  if (ty < WS % SOW)
  {
    *g_inout = **bdata;
  }
}

/**
 *  @ingroup gpu
 *  @overload
 *  @brief Algorithm SAT stage 4 (not-in-place computation)
 *
 *  @note The CUDA kernel functions (as this one) have many
 *  idiosyncrasies and should not be used lightly.
 *
 *  @see [Nehab:2011] cited in alg5() and figure in algSAT()
 *  @param[out] g_out The output image
 *  @param[in] g_in The input image
 *  @param[in] g_y All \f$P_{m,n}(Y)\f$
 *  @param[in] g_v All \f$P^T_{m,n}(V)\f$
 */
__global__ void algSAT_stage4(const int c_width, const int c_height,
                              float *g_out, const float *g_in, const float *g_y,
                              const float *g_v)
{
  const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x,
            by = blockIdx.y, col = bx * WS + tx, row0 = by * WS;

  __shared__ float s_block[WS][WS + 1];

  float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[ty][tx];

  g_in += (row0 + ty) * c_width + col;
  if (by > 0)
    g_y += (by - 1) * c_width + col;
  if (bx > 0)
    g_v += (bx - 1) * c_height + row0 + tx;

#pragma unroll
  for (int i = 0; i < WS - (WS % SOW); i += SOW)
  {
    **bdata = *g_in;
    bdata += SOW;
    g_in += SOW * c_width;
  }
  if (ty < WS % SOW)
  {
    **bdata = *g_in;
  }

  __syncthreads();

  if (ty == 0)
  {
    {  // calculate y -----------------------
      float(*bdata)[WS + 1] = (float(*)[WS + 1]) & s_block[0][tx];

      float prev;
      if (by > 0)
        prev = *g_y;
      else
        prev = 0.f;

#pragma unroll
      for (int i = 0; i < WS; ++i, ++bdata)
        **bdata = prev = **bdata + prev;
    }

    {  // calculate x -----------------------
      float *bdata = s_block[tx];

      float prev;
      if (bx > 0)
        prev = *g_v;
      else
        prev = 0.f;

#pragma unroll
      for (int i = 0; i < WS; ++i, ++bdata)
        *bdata = prev = *bdata + prev;
    }
  }

  __syncthreads();

  bdata = (float(*)[WS + 1]) & s_block[ty][tx];

  g_out += (row0 + ty) * c_width + col;

#pragma unroll
  for (int i = 0; i < WS - (WS % SOW); i += SOW)
  {
    *g_out = **bdata;
    bdata += SOW;
    g_out += SOW * c_width;
  }
  if (ty < WS % SOW)
  {
    *g_out = **bdata;
  }
}

/*
__host__
void prepare_algSAT(const int width
    dvector<float>& d_inout,
    dvector<float>& d_ybar,
    dvector<float>& d_vhat,
    dvector<float>& d_ysum,
    const float *h_in,
    const int& w,
    const int& h ) {

  algs.width = w;
  algs.height = h;

  if( w % 32 > 0 ) algs.width += (32 - (w % 32));
  if( h % 32 > 0 ) algs.height += (32 - (h % 32));

  calc_alg_setup( algs, algs.width, algs.height );
  up_alg_setup( algs );

  d_inout.copy_from( h_in, w, h, algs.width, algs.height );

  d_ybar.resize( algs.n_size * algs.width );
  d_vhat.resize( algs.m_size * algs.height );
  d_ysum.resize( algs.m_size * algs.n_size );

}
*/

__global__ void sat_init_kernel(int image_size, float xscale, float yscale,
                                float *thrustptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * image_size + x;

  float texval = tex2D(textureReadDepth, x * xscale + 0.5f, y * yscale + 0.5f);

  thrustptr[index] = texval;
}

void resizeIfNecessary(thrust::device_vector<float> &vector, unsigned long size)
{
  if (vector.size() != size)
  {
    vector.resize(size);
  }
}

void callStages(thrust::device_vector<float> &inout,
                thrust::device_vector<float> &ybar,
                thrust::device_vector<float> &vhat,
                thrust::device_vector<float> &ysum, int computeWidth,
                int computeHeight, int compute_m_size, int compute_n_size)
{
  const int nWm = (computeWidth + MTS - 1) / MTS,
            nHm = (computeHeight + MTS - 1) / MTS;
  const dim3 cg_img(compute_m_size, compute_n_size);
  const dim3 cg_ybar(nWm, 1);
  const dim3 cg_vhat(1, nHm);

  float *d_inout = thrust::raw_pointer_cast(inout.data());
  float *d_ybar = thrust::raw_pointer_cast(ybar.data());
  float *d_vhat = thrust::raw_pointer_cast(vhat.data());
  float *d_ysum = thrust::raw_pointer_cast(ysum.data());

  algSAT_stage1<<<cg_img, dim3(WS, SOW)>>>(computeWidth, computeHeight,
      d_inout, d_ybar, d_vhat);
  algSAT_stage2<<<cg_ybar, dim3(WS, MW)>>>(compute_m_size, compute_n_size,
      computeWidth, d_ybar, d_ysum);
  algSAT_stage3<<<cg_vhat, dim3(WS, MW)>>>(compute_m_size, computeHeight,
      d_ysum, d_vhat);
  algSAT_stage4<<<cg_img, dim3(WS, SOW)>>>(computeWidth, computeHeight,
      d_inout, d_ybar, d_vhat);
}

void cudaSAT(hipGraphicsResource_t &inputImage, int image_size,
             int screen_size_x, int screen_size_y, float z_threshold,
             thrust::device_vector<float> &inout,
             thrust::device_vector<float> &ybar,
             thrust::device_vector<float> &vhat,
             thrust::device_vector<float> &ysum)
{
  int computeWidth = image_size;
  int computeHeight = image_size;
  if (computeWidth % 32 > 0)
    computeWidth += (32 - (computeWidth % 32));
  if (computeHeight % 32 > 0)
    computeHeight += (32 - (computeHeight % 32));
  int compute_m_size = (computeWidth + WS - 1) / WS;
  int compute_n_size = (computeHeight + WS - 1) / WS;

  // set data structure sizes
  resizeIfNecessary(inout, computeWidth * computeHeight);
  resizeIfNecessary(ybar, compute_n_size * computeWidth);
  resizeIfNecessary(vhat, compute_m_size * computeHeight);
  resizeIfNecessary(ysum, compute_m_size * compute_n_size);


  // QElapsedTimer tm;
  // tm.start();

  // initialize occupancy function from inputImage
  textureReadDepth.normalized = 0;
  textureReadDepth.filterMode = hipFilterModeLinear /*hipFilterModePoint*/;
  textureReadDepth.addressMode[0] = hipAddressModeWrap;
  textureReadDepth.addressMode[1] = hipAddressModeWrap;

  hipGraphicsMapResources(1, &inputImage);
  hipArray_t input_array;
  hipGraphicsSubResourceGetMappedArray(&input_array, inputImage, 0, 0);
  hipChannelFormatDesc channeldesc;
  hipGetChannelDesc(&channeldesc, input_array);

  hipBindTextureToArray(&textureReadDepth, input_array, &channeldesc);

  dim3 dimBlock(64, 1, 1);
  dim3 dimGrid(divUp(image_size, dimBlock.x), divUp(image_size, dimBlock.y), 1);

  float *d_inout = thrust::raw_pointer_cast(inout.data());
  sat_init_kernel << <dimGrid, dimBlock>>>(image_size,
       static_cast<float>(screen_size_x) / static_cast<float>(image_size),
       static_cast<float>(screen_size_y) / static_cast<float>(image_size),
       d_inout);
  hipDeviceSynchronize();
  hipUnbindTexture(&textureReadDepth);
  hipGraphicsUnmapResources(1, &inputImage);

  callStages(inout, ybar, vhat, ysum, computeWidth, computeHeight,
             compute_m_size, compute_n_size);
}

/*
__host__
void algSAT( dvector<float>& d_out,
    dvector<float>& d_ybar,
    dvector<float>& d_vhat,
    dvector<float>& d_ysum,
    const dvector<float>& d_in,
    const alg_setup& algs ) {

  const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
  const dim3 cg_img( algs.m_size, algs.n_size );
  const dim3 cg_ybar( nWm, 1 );
  const dim3 cg_vhat( 1, nHm );

  algSAT_stage1<<< cg_img, dim3(WS, SOW) >>>( d_in, d_ybar, d_vhat );

  algSAT_stage2<<< cg_ybar, dim3(WS, MW) >>>( d_ybar, d_ysum );

  algSAT_stage3<<< cg_vhat, dim3(WS, MW) >>>( d_ysum, d_vhat );

  algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_out, d_in, d_ybar, d_vhat );

}

__host__
void algSAT( dvector<float>& d_inout,
    dvector<float>& d_ybar,
    dvector<float>& d_vhat,
    dvector<float>& d_ysum,
    const alg_setup& algs ) {

  const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
  const dim3 cg_img( algs.m_size, algs.n_size );
  const dim3 cg_ybar( nWm, 1 );
  const dim3 cg_vhat( 1, nHm );

  algSAT_stage1<<< cg_img, dim3(WS, SOW) >>>( d_inout, d_ybar, d_vhat );

  algSAT_stage2<<< cg_ybar, dim3(WS, MW) >>>( d_ybar, d_ysum );

  algSAT_stage3<<< cg_vhat, dim3(WS, MW) >>>( d_ysum, d_vhat );

  algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_inout, d_ybar, d_vhat );

}
*/

thrust::host_vector<float> algSAT(float *h_inout, int w, int h)
{
  int computeWidth = w;
  int computeHeight = w;
  if (computeWidth % 32 > 0)
    computeWidth += (32 - (computeWidth % 32));
  if (computeHeight % 32 > 0)
    computeHeight += (32 - (computeHeight % 32));
  int compute_m_size = (computeWidth + WS - 1) / WS;
  int compute_n_size = (computeHeight + WS - 1) / WS;

  // alg_setup algs;
  thrust::device_vector<float> inout(h_inout, h_inout + w * h);
  thrust::device_vector<float> ybar;
  thrust::device_vector<float> vhat;
  thrust::device_vector<float> ysum;

  // prepare_algSAT( algs, d_out, d_ybar, d_vhat, d_ysum, h_inout, w, h );
  resizeIfNecessary(inout, computeWidth * computeHeight);
  resizeIfNecessary(ybar, compute_n_size * computeWidth);
  resizeIfNecessary(vhat, compute_m_size * computeHeight);
  resizeIfNecessary(ysum, compute_m_size * compute_n_size);

  callStages(inout, ybar, vhat, ysum, computeWidth, computeHeight,
             compute_m_size, compute_n_size);

  hipDeviceSynchronize();
  // algSAT(d_out, d_ybar, d_vhat, d_ysum, algs);

  // thrust::host_vector<float> result = inout;
  // thrust::host_vector<float> result(w * h);
  // thrust::copy(inout.begin(), inout.end(), result.begin());

  return inout;
}

namespace Placement
{

SummedAreaTable::SummedAreaTable(std::shared_ptr<CudaArrayProvider> inputImage)
  : inputImage(inputImage)
{
}

void SummedAreaTable::runKernel()
{
  int computeWidth = inputImage->getWidth();
  int computeHeight = inputImage->getHeight();
  if (computeWidth % 32 > 0)
    computeWidth += (32 - (computeWidth % 32));
  if (computeHeight % 32 > 0)
    computeHeight += (32 - (computeHeight % 32));
  int compute_m_size = (computeWidth + WS - 1) / WS;
  int compute_n_size = (computeHeight + WS - 1) / WS;

  std::cout << computeWidth << "/" << computeHeight << std::endl;

  // set data structure sizes
  resizeIfNecessary(inout, computeWidth * computeHeight);
  resizeIfNecessary(ybar, compute_n_size * computeWidth);
  resizeIfNecessary(vhat, compute_m_size * computeHeight);
  resizeIfNecessary(ysum, compute_m_size * compute_n_size);

  // initialize occupancy function from inputImage
  textureReadDepth.normalized = 0;
  textureReadDepth.filterMode = hipFilterModeLinear /*hipFilterModePoint*/;
  textureReadDepth.addressMode[0] = hipAddressModeWrap;
  textureReadDepth.addressMode[1] = hipAddressModeWrap;

  inputImage->map();
  hipBindTextureToArray(textureReadDepth, inputImage->getArray(),
                         inputImage->getChannelDesc());

  dim3 dimBlock(64, 1, 1);
  dim3 dimGrid(divUp(inputImage->getWidth(), dimBlock.x),
               divUp(inputImage->getHeight(), dimBlock.y), 1);

  float *d_inout = thrust::raw_pointer_cast(inout.data());
  int image_size = inputImage->getWidth();
  int screen_size_x = inputImage->getWidth();
  int screen_size_y = inputImage->getHeight();

  sat_init_kernel<<<dimGrid, dimBlock>>>(image_size,
       static_cast<float>(screen_size_x) / static_cast<float>(image_size),
       static_cast<float>(screen_size_y) / static_cast<float>(image_size),
       d_inout);
  hipDeviceSynchronize();
  hipUnbindTexture(&textureReadDepth);
  inputImage->unmap();

  callStages(inout, ybar, vhat, ysum, computeWidth, computeHeight,
             compute_m_size, compute_n_size);
}

thrust::device_vector<float> &SummedAreaTable::getResults()
{
  return inout;
}

}  // namespace Placement

#include "hip/hip_runtime.h"
#include "./direct_integral_costs_calculator.h"
#include "../utils/cuda_helper.h"

__global__ void integralCosts(hipTextureObject_t colors,
                                 float occlusionWeight,
                                 hipTextureObject_t saliency,
                                 float saliencyWeight,
                                 hipSurfaceObject_t output, int width,
                                 int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  surf2Dwrite(1.0f, output, x * sizeof(float), y);
}


namespace Placement
{

DirectIntegralCostsCalculator::DirectIntegralCostsCalculator(
    std::shared_ptr<CudaArrayProvider> colorProvider,
    std::shared_ptr<CudaArrayProvider> saliencyProvider,
    std::shared_ptr<CudaArrayProvider> outputProvider)
  : colorProvider(colorProvider), saliencyProvider(saliencyProvider),
    outputProvider(outputProvider)
{
}

DirectIntegralCostsCalculator::~DirectIntegralCostsCalculator()
{
  if (color)
    hipDestroyTextureObject(color);
  if (saliency)
    hipDestroyTextureObject(saliency);
  if (output)
    hipDestroySurfaceObject(output);
}

void DirectIntegralCostsCalculator::runKernel()
{
  if (!color)
    createSurfaceObjects();

  int outputWidth = outputProvider->getWidth();
  int outputHeight = outputProvider->getHeight();

  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid(divUp(outputWidth, dimBlock.x), divUp(outputHeight, dimBlock.y),
               1);

  integralCosts<<<dimGrid, dimBlock>>>(color, weights.occlusion,
                                          saliency, weights.saliency, output,
                                          outputWidth, outputHeight);

  HANDLE_ERROR(hipDeviceSynchronize());
}

void DirectIntegralCostsCalculator::createSurfaceObjects()
{
  colorProvider->map();
  saliencyProvider->map();
  outputProvider->map();

  auto resDesc = colorProvider->getResourceDesc();
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&color, &resDesc, &texDesc, NULL);

  auto saliencyResDesc = saliencyProvider->getResourceDesc();
  hipCreateTextureObject(&saliency, &saliencyResDesc, &texDesc, NULL);

  auto outputResDesc = outputProvider->getResourceDesc();
  hipCreateSurfaceObject(&output, &outputResDesc);

  colorProvider->unmap();
  saliencyProvider->unmap();
  outputProvider->unmap();
}

}  // namespace Placement

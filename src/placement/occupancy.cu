#include "hip/hip_runtime.h"
#include "./occupancy.h"
#include "../utils/cuda_helper.h"

__global__ void occupancyKernel(hipTextureObject_t positions,
    hipSurfaceObject_t output, int width, int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  float4 position = tex2D<float4>(positions, x + 0.5f, y + 0.5f);

  float outputValue = position.z;
  surf2Dwrite(outputValue, output, x * sizeof(float), y);
}

Occupancy::Occupancy(std::shared_ptr<CudaArrayProvider> positionProvider,
                     std::shared_ptr<CudaArrayProvider> outputProvider)
  : positionProvider(positionProvider), outputProvider(outputProvider)
{
  createSurfaceObjects();
}

Occupancy::~Occupancy()
{
  if (positions)
    hipDestroyTextureObject(positions);
  if (output)
    hipDestroySurfaceObject(output);
}

void Occupancy::runKernel()
{
  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid(divUp(positionProvider->getWidth(), dimBlock.x),
               divUp(positionProvider->getHeight(), dimBlock.y), 1);

  occupancyKernel<<<dimGrid, dimBlock>>>(positions, output,
      positionProvider->getWidth(), positionProvider->getHeight());
  HANDLE_ERROR(hipDeviceSynchronize());

}

void Occupancy::createSurfaceObjects()
{
  positionProvider->map();
  outputProvider->map();

  auto resDesc = positionProvider->getResourceDesc();
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&positions, &resDesc, &texDesc, NULL);

  auto outputResDesc = outputProvider->getResourceDesc();
  hipCreateSurfaceObject(&output, &outputResDesc);

  positionProvider->unmap();
  outputProvider->unmap();
}


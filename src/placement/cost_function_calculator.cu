#include "hip/hip_runtime.h"
#include "./cost_function_calculator.h"
#include <thrust/transform_reduce.h>
#include <limits>

struct EvalResult
{
  __host__ __device__ EvalResult()
  {
  }

  __host__ __device__ EvalResult(int x, int y, float cost)
    : x(x), y(y), cost(cost)
  {
  }

  int x, y;

  float cost;

  bool operator<(const EvalResult &other)
  {
    return thrust::min<float>(this->cost, other.cost);
  }
};

__host__ __device__ bool operator<(const EvalResult &a, const EvalResult &b)
{
  return (a.cost < b.cost);
}

struct CostEvaluator : public thrust::unary_function<int, EvalResult>
{
  __host__ __device__ CostEvaluator(int width, int height)
    : width(width), height(height)
  {
  }

  int width;
  int height;

  int halfLabelWidth;
  int halfLabelHeight;

  float anchorX;
  float anchorY;

  const float *occupancy;

  __device__ float lineLength(int x, int y) const
  {
    float diffX = x - anchorX;
    float diffY = y - anchorY;

    return sqrt(diffX * diffX + diffY * diffY);
  }

  __device__ float favorHorizontalOrVerticalLines(
      int x, int y) const
  {
    float diffX = x - anchorX;
    float diffY = y - anchorY;

    float length = sqrt(diffX * diffX + diffY * diffY);
    diffX = diffX / length;
    diffY = diffY / length;

    return fabs(diffX) + fabs(diffY);
  }

  __device__ float occupancyForLabelArea(int x, int y) const
  {
    int startX = max(x - halfLabelWidth, 0);
    int startY = max(y - halfLabelWidth, 0);
    int endX = min(x + halfLabelWidth, width - 1);
    int endY = min(y + halfLabelWidth, height - 1);

    float sum =
        occupancy[endY * width + endX] - occupancy[startY * width + startX];

    return sum / (4 * halfLabelWidth * halfLabelHeight);
  }

  __device__ EvalResult operator()(const int &index) const
  {
    int x = index % width;
    int y = index / width;

    float distanceToAnchor = lineLength(x, y);

    float cost = distanceToAnchor + 10.0f * occupancyForLabelArea(x, y) +
                 favorHorizontalOrVerticalLines(x, y);
    EvalResult result(x, y, cost);

    return result;
  }
};

template <typename T>
struct MinimumCostOperator : public thrust::binary_function<T, T, T>
{
  __host__ __device__ T operator()(const T &x, const T &y) const
  {
    T result;

    result = x < y ? x : y;
    return result;
  }
};

void CostFunctionCalculator::resize(int width, int height)
{
  this->width = width;
  this->height = height;
}

void CostFunctionCalculator::calculateCosts(
    const thrust::device_vector<float> &distances)
{
  // calculateForLabel(distances, 0, 500, 500);
}

std::tuple<float, float> CostFunctionCalculator::calculateForLabel(
    const thrust::device_vector<float> &occupancySummedAreaTable, int labelId,
    float anchorX, float anchorY, int sizeX, int sizeY)
{
  CostEvaluator costEvaluator(width, height);
  costEvaluator.anchorX = anchorX;
  costEvaluator.anchorY = anchorY;
  costEvaluator.occupancy = thrust::raw_pointer_cast(occupancySummedAreaTable.data());
  costEvaluator.halfLabelWidth = sizeX / 2;
  costEvaluator.halfLabelHeight = sizeY / 2;

  MinimumCostOperator<EvalResult> minimumCostOperator;
  EvalResult initialCost;
  initialCost.x = -1;
  initialCost.y = -1;
  initialCost.cost = std::numeric_limits<float>::max();

  EvalResult cost = thrust::transform_reduce(
      thrust::counting_iterator<int>(0),
      thrust::counting_iterator<int>(0) + width * height, costEvaluator,
      initialCost, minimumCostOperator);

  std::cout << cost.x << "/" << cost.y << ": " << cost.cost << std::endl;
  return std::make_tuple(cost.x, cost.y);
}


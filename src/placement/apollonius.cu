#include "hip/hip_runtime.h"
#include "./apollonius.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "../utils/cuda_helper.h"

__global__ void seed(hipSurfaceObject_t output, int imageSize, int labelCount,
                     float4 *seedbuffer, int *thrustptr, int *idptr,
                     int *idxptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;

  int index = y * imageSize + x;
  float4 outval = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
  float4 seedval = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

  // initialize to out of bounds
  int outindex = (imageSize * 2) * (imageSize * 2) - 1;

  for (int i = 0; i < labelCount; i++)
  {
    float4 seedval = seedbuffer[i];
    int4 seedValueInt =
        make_int4(static_cast<int>(seedval.x), static_cast<int>(seedval.y),
                  static_cast<int>(seedval.z), static_cast<int>(seedval.w));
    if (seedValueInt.x > 0 && x == seedValueInt.y && y == seedValueInt.z &&
        (x != 0 || y != 0))
    {
      outval =
          make_float4(seedval.x / (labelCount + 1),
                      seedValueInt.y / static_cast<float>(imageSize),
                      seedValueInt.z / static_cast<float>(imageSize), 1.0f);

      outindex = x + y * imageSize;
    }
    idptr[i] = seedValueInt.x;
    idxptr[i] = seedValueInt.y + seedValueInt.z * imageSize;
  }

  thrustptr[index] = outindex;
  surf2Dwrite(outval, output, x * sizeof(float4), y);
}

__global__ void apolloniusStep(int *data, float *occupancy, unsigned int step,
                               int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= w || y >= h)
    return;

  int index = y * w + x;

  int currentNearest = data[index];
  int currentY = currentNearest / w;
  int currentX = currentNearest - currentY * w;
  float curr_w = (currentNearest < w * h) ? occupancy[currentNearest] : 0.0f;

  float currentDistance =
      sqrtf(static_cast<float>((x - currentX) * (x - currentX) +
                               (y - currentY) * (y - currentY))) -
      curr_w;

#pragma unroll
  for (int i = -1; i <= 1; i++)
  {
    int u = x + i * step;
    if (u < 0 || u >= w)
      continue;
#pragma unroll
    for (int j = -1; j <= 1; j += 2 - i * i)
    {
      int v = y + j * step;
      if (v < 0 || v >= h)
        continue;

      int newindex = v * w + u;
      int newNearest = data[newindex];
      int newY = newNearest / w;
      int newX = newNearest - newY * w;
      float newW = (newNearest < w * h) ? occupancy[newNearest] : 0.0f;
      float newDistance = sqrtf(static_cast<float>((x - newX) * (x - newX) +
                                                   (y - newY) * (y - newY))) -
                          newW;

      if (newDistance < currentDistance || currentNearest >= w * h)
      {
        currentDistance = newDistance;
        currentNearest = newNearest;
      }
    }
  }

  data[index] = currentNearest;
}

__global__ void gather(hipSurfaceObject_t output, int imageSize,
                       int labelCount, int *thrustptr, int *seedidptr,
                       int *seedidxptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;
  int index = y * imageSize + x;
  float4 color;
  int labelID = 100;
  int labelIndex = thrustptr[index];

  for (int i = 0; i < labelCount; i++)
  {
    if (labelIndex == seedidxptr[i])
    {
      labelID = seedidptr[i];
      break;
    }
  }

  switch (labelID)
  {
  case 0:
    color = make_float4(0.0, 0.0, 0.0, 1.0);
    break;
  case 1:
    color = make_float4(1.0, 0.0, 0.0, 1.0);
    break;
  case 2:
    color = make_float4(0.0, 1.0, 0.0, 1.0);
    break;
  case 3:
    color = make_float4(0.0, 0.0, 1.0, 1.0);
    break;
  case 4:
    color = make_float4(1.0, 1.0, 0.0, 1.0);
    break;
  case 5:
    color = make_float4(0.0, 1.0, 1.0, 1.0);
    break;
  case 6:
    color = make_float4(1.0, 0.0, 1.0, 1.0);
    break;
  case 7:
    color = make_float4(1.0, 1.0, 1.0, 1.0);
    break;
  default:
    color = make_float4(0.5, 0.5, 0.5, 1.0);
  }
  surf2Dwrite(color, output, x * sizeof(float4), y);
}

Apollonius::Apollonius(std::shared_ptr<CudaArrayProvider> inputImage,
                       thrust::device_vector<float4> &seedBuffer,
                       thrust::device_vector<float> &distances, int labelCount)
  : inputImage(inputImage), seedBuffer(seedBuffer), distances(distances),
    labelCount(labelCount)
{
  imageSize = inputImage->getWidth();
  pixelCount = imageSize * imageSize;
}

void Apollonius::run()
{
  resize();
  inputImage->map();
  auto resDesc = inputImage->getResourceDesc();
  hipCreateSurfaceObject(&outputSurface, &resDesc);

  dimBlock = dim3(32, 32, 1);
  dimGrid = dim3(divUp(imageSize, dimBlock.x), divUp(imageSize, dimBlock.y), 1);

  runSeedKernel();
  runStepsKernels();
  runGatherKernel();

  inputImage->unmap();
}

void Apollonius::resize()
{
  if (computeVector.size() != static_cast<unsigned long>(pixelCount))
  {
    computeVector.resize(pixelCount, pixelCount);
    computeVectorTemp.resize(pixelCount, pixelCount);
  }

  if (seedIds.size() != MAX_LABELS || seedIndices.size() != MAX_LABELS)
  {
    seedIds.resize(MAX_LABELS, -1);
    seedIndices.resize(MAX_LABELS, -1);
  }
}

void Apollonius::runSeedKernel()
{
  int *raw_ptr = thrust::raw_pointer_cast(computeVector.data());
  int *idptr = thrust::raw_pointer_cast(seedIds.data());
  int *idxptr = thrust::raw_pointer_cast(seedIndices.data());
  float4 *seedBufferPtr = thrust::raw_pointer_cast(seedBuffer.data());

  seed<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount, seedBufferPtr, raw_ptr,
      idptr, idxptr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runStepsKernels()
{
  computeVectorTemp = computeVector;
  apolloniusStep<<<dimGrid, dimBlock>>>
      (thrust::raw_pointer_cast(computeVector.data()),
       thrust::raw_pointer_cast(distances.data()), 1, imageSize,
       imageSize);

  for (int k = (imageSize / 2); k > 0; k /= 2)
  {
    apolloniusStep<<<dimGrid, dimBlock>>>(
        thrust::raw_pointer_cast(computeVector.data()),
        thrust::raw_pointer_cast(distances.data()), k, imageSize,
        imageSize);
  }
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runGatherKernel()
{
  int *raw_ptr = thrust::raw_pointer_cast(computeVector.data());
  int *idptr = thrust::raw_pointer_cast(seedIds.data());
  int *idxptr = thrust::raw_pointer_cast(seedIndices.data());
  gather<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount, raw_ptr,
      idptr, idxptr);
  HANDLE_ERROR(hipDeviceSynchronize());
}


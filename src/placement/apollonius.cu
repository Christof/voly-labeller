#include "hip/hip_runtime.h"
#include "./apollonius.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "../utils/cuda_helper.h"

__global__ void seed(hipSurfaceObject_t output, int imageSize, int labelCount,
                     float4 *seedBuffer, int *computePtr, int *idPtr,
                     int *indicesPtr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;

  int index = y * imageSize + x;
  float4 outValue = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

  // initialize to out of bounds
  int outIndex = (imageSize * 2) * (imageSize * 2) - 1;

  for (int i = 0; i < labelCount; i++)
  {
    float4 seedValue = seedBuffer[i];
    int4 seedValueInt =
        make_int4(static_cast<int>(seedValue.x), static_cast<int>(seedValue.y),
                  static_cast<int>(seedValue.z), static_cast<int>(seedValue.w));
    if (seedValueInt.x > 0 && x == seedValueInt.y && y == seedValueInt.z &&
        (x != 0 || y != 0))
    {
      outValue =
          make_float4(seedValue.x / (labelCount + 1),
                      seedValueInt.y / static_cast<float>(imageSize),
                      seedValueInt.z / static_cast<float>(imageSize), 1.0f);

      outIndex = x + y * imageSize;
    }
    idPtr[i] = seedValueInt.x;
    indicesPtr[i] = seedValueInt.y + seedValueInt.z * imageSize;
  }

  computePtr[index] = outIndex;
  surf2Dwrite(outValue, output, x * sizeof(float4), y);
}

__global__ void apolloniusStep(int *data, float *occupancy, unsigned int step,
                               int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= w || y >= h)
    return;

  int index = y * w + x;

  int currentNearest = data[index];
  int currentY = currentNearest / w;
  int currentX = currentNearest - currentY * w;
  float curr_w = (currentNearest < w * h) ? occupancy[currentNearest] : 0.0f;

  float currentDistance =
      sqrtf(static_cast<float>((x - currentX) * (x - currentX) +
                               (y - currentY) * (y - currentY))) -
      curr_w;

#pragma unroll
  for (int i = -1; i <= 1; i++)
  {
    int u = x + i * step;
    if (u < 0 || u >= w)
      continue;
#pragma unroll
    for (int j = -1; j <= 1; j += 2 - i * i)
    {
      int v = y + j * step;
      if (v < 0 || v >= h)
        continue;

      int newindex = v * w + u;
      int newNearest = data[newindex];
      int newY = newNearest / w;
      int newX = newNearest - newY * w;
      float newW = (newNearest < w * h) ? occupancy[newNearest] : 0.0f;
      float newDistance = sqrtf(static_cast<float>((x - newX) * (x - newX) +
                                                   (y - newY) * (y - newY))) -
                          newW;

      if (newDistance < currentDistance || currentNearest >= w * h)
      {
        currentDistance = newDistance;
        currentNearest = newNearest;
      }
    }
  }

  data[index] = currentNearest;
}

__global__ void gather(hipSurfaceObject_t output, int imageSize,
                       int labelCount, int *nearestIndex, int *seedIds,
                       int *seedIndices)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;

  int index = y * imageSize + x;
  float4 color;
  int labelId = 100;
  int labelIndex = nearestIndex[index];

  for (int i = 0; i < labelCount; i++)
  {
    if (labelIndex == seedIndices[i])
    {
      labelId = seedIds[i];
      break;
    }
  }

  switch (labelId)
  {
  case 0:
    color = make_float4(0.0, 0.0, 0.0, 1.0);
    break;
  case 1:
    color = make_float4(1.0, 0.0, 0.0, 1.0);
    break;
  case 2:
    color = make_float4(0.0, 1.0, 0.0, 1.0);
    break;
  case 3:
    color = make_float4(0.0, 0.0, 1.0, 1.0);
    break;
  case 4:
    color = make_float4(1.0, 1.0, 0.0, 1.0);
    break;
  case 5:
    color = make_float4(0.0, 1.0, 1.0, 1.0);
    break;
  case 6:
    color = make_float4(1.0, 0.0, 1.0, 1.0);
    break;
  case 7:
    color = make_float4(1.0, 1.0, 1.0, 1.0);
    break;
  default:
    color = make_float4(0.5, 0.5, 0.5, 1.0);
  }
  surf2Dwrite(color, output, x * sizeof(float4), y);
}

Apollonius::Apollonius(std::shared_ptr<CudaArrayProvider> inputImage,
                       thrust::device_vector<float4> &seedBuffer,
                       thrust::device_vector<float> &distances, int labelCount)
  : inputImage(inputImage), seedBuffer(seedBuffer), distances(distances),
    labelCount(labelCount)
{
  imageSize = inputImage->getWidth();
  pixelCount = imageSize * imageSize;
}

void Apollonius::run()
{
  resize();
  inputImage->map();
  auto resDesc = inputImage->getResourceDesc();
  hipCreateSurfaceObject(&outputSurface, &resDesc);

  dimBlock = dim3(32, 32, 1);
  dimGrid = dim3(divUp(imageSize, dimBlock.x), divUp(imageSize, dimBlock.y), 1);

  runSeedKernel();
  runStepsKernels();
  runGatherKernel();

  inputImage->unmap();
}

thrust::device_vector<int> &Apollonius::getIds()
{
  return seedIds;
}

void Apollonius::resize()
{
  if (computeVector.size() != static_cast<unsigned long>(pixelCount))
  {
    computeVector.resize(pixelCount, pixelCount);
  }

  if (seedIds.size() != MAX_LABELS || seedIndices.size() != MAX_LABELS)
  {
    seedIds.resize(MAX_LABELS, -1);
    seedIndices.resize(MAX_LABELS, -1);
  }
}

void Apollonius::runSeedKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  int *idPtr = thrust::raw_pointer_cast(seedIds.data());
  int *indicesPtr = thrust::raw_pointer_cast(seedIndices.data());
  float4 *seedBufferPtr = thrust::raw_pointer_cast(seedBuffer.data());

  seed<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount,
                              seedBufferPtr, computePtr, idPtr, indicesPtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runStepsKernels()
{
  apolloniusStep<<<dimGrid, dimBlock>>>
      (thrust::raw_pointer_cast(computeVector.data()),
       thrust::raw_pointer_cast(distances.data()), 1, imageSize, imageSize);

  for (int k = (imageSize / 2); k > 0; k /= 2)
  {
    apolloniusStep<<<dimGrid, dimBlock>>>
        (thrust::raw_pointer_cast(computeVector.data()),
         thrust::raw_pointer_cast(distances.data()), k, imageSize, imageSize);
  }
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runGatherKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  int *seedIdsPtr = thrust::raw_pointer_cast(seedIds.data());
  int *seedIndicesPtr = thrust::raw_pointer_cast(seedIndices.data());
  gather<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount,
      computePtr, seedIdsPtr, seedIndicesPtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

thrust::device_vector<float4>
Apollonius::createSeedBufferFromLabels(std::vector<Label> labels,
                                       Eigen::Matrix4f viewProjection,
                                       Eigen::Vector2i size)
{
  thrust::host_vector<float4> result;
  for (auto &label : labels)
  {
    Eigen::Vector4f pos =
        viewProjection * Eigen::Vector4f(label.anchorPosition.x(),
                                         label.anchorPosition.y(),
                                         label.anchorPosition.z(), 1);
    float x = (pos.x() / pos.w() * 0.5f + 0.5f) * size.x();
    float y = (pos.y() / pos.w() * 0.5f + 0.5f) * size.y();
    result.push_back(make_float4(label.id, x, y, 1));
  }

  return result;
}


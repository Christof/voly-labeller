#include "hip/hip_runtime.h"
#include "./apollonius.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <vector>
#include "../utils/cuda_helper.h"

__global__ void seed(hipSurfaceObject_t output, int imageSize, int labelCount,
                     float4 *seedBuffer, int *computePtr, int *idPtr,
                     int *indicesPtr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;

  int index = y * imageSize + x;

  // initialize to out of bounds
  int outIndex = (imageSize * 2) * (imageSize * 2) - 1;

  for (int i = 0; i < labelCount; i++)
  {
    float4 seedValue = seedBuffer[i];
    int4 seedValueInt =
        make_int4(static_cast<int>(seedValue.x), static_cast<int>(seedValue.y),
                  static_cast<int>(seedValue.z), static_cast<int>(seedValue.w));
    if (seedValueInt.x > 0 && x == seedValueInt.y && y == seedValueInt.z &&
        (x != 0 || y != 0))
    {
      outIndex = x + y * imageSize;
    }

    idPtr[i] = seedValueInt.x;
    indicesPtr[i] = seedValueInt.y + seedValueInt.z * imageSize;
  }

  computePtr[index] = outIndex;
}

__global__ void apolloniusStep(hipTextureObject_t distances, int *data,
                               unsigned int step, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= w || y >= h)
    return;

  int index = y * w + x;

  int currentNearest = data[index];
  int currentY = currentNearest / w;
  int currentX = currentNearest - currentY * w;
  float curr_w = (currentNearest < w * h)
                     ? tex2D<float>(distances, currentX + 0.5f, currentY + 0.5f)
                     : 0.0f;

  float currentDistance =
      sqrtf(static_cast<float>((x - currentX) * (x - currentX) +
                               (y - currentY) * (y - currentY))) -
      curr_w;

#pragma unroll
  for (int i = -1; i <= 1; i++)
  {
    int u = x + i * step;
    if (u < 0 || u >= w)
      continue;
#pragma unroll
    for (int j = -1; j <= 1; j += 2 - i * i)
    {
      int v = y + j * step;
      if (v < 0 || v >= h)
        continue;

      int newindex = v * w + u;
      int newNearest = data[newindex];
      int newY = newNearest / w;
      int newX = newNearest - newY * w;
      float newW = (newNearest < w * h)
                       ? tex2D<float>(distances, newX + 0.5f, newY + 0.5f)
                       : 0.0f;
      float newDistance = sqrtf(static_cast<float>((x - newX) * (x - newX) +
                                                   (y - newY) * (y - newY))) -
                          newW;

      if (newDistance < currentDistance || currentNearest >= w * h)
      {
        currentDistance = newDistance;
        currentNearest = newNearest;
      }
    }
  }

  data[index] = currentNearest;
}

/**
 * Taken from http://stackoverflow.com/questions/20792445/calculate-rgb-value-for-a-range-of-values-to-create-heat-map
 */
__device__ float4 rgb(int maxValue, int value)
{
    float ratio = 2.0f * value / maxValue;
    float b = max(0.0f, 1 - ratio);
    float r = max(0.0f, ratio - 1);
    float g = max(0.0f, 1.0f - b - r);

    return make_float4(r, g, b, 1.0);
}

__global__ void gather(hipSurfaceObject_t output, int imageSize,
                       int labelCount, int *nearestIndex, int *seedIds,
                       int *seedIndices)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imageSize || y >= imageSize)
    return;

  int index = y * imageSize + x;
  int labelId = -1;
  int labelIndex = nearestIndex[index];

  for (int i = 0; i < labelCount; i++)
  {
    if (labelIndex == seedIndices[i])
    {
      labelId = seedIds[i];
      break;
    }
  }

  float4 color = labelId == -1 ?
    make_float4(1, 1, 1, 1) : rgb(labelCount, labelId);

  surf2Dwrite(color, output, x * sizeof(float4), y);
}

__global__ void copyBorderIndex(int imageSize, int *source, int *destination)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int maxIndex = imageSize * imageSize - 1;

  // FIXME: corner pixels are duplicated
  if (index >= imageSize)
    return;

  // upper border from left to right
  destination[index] = source[index];

  // right border from top to bottom
  destination[imageSize + index] =
      source[(imageSize - 1) + index * imageSize];

  // bottom border from right to left
  destination[imageSize * 2 + index] = source[maxIndex - index];

  // left border from bottom to top
  destination[imageSize * 3 + index] =
      source[maxIndex - imageSize + 1 - index * imageSize];
}

namespace Placement
{

Apollonius::Apollonius(std::shared_ptr<CudaArrayProvider> distancesImage,
                       std::shared_ptr<CudaArrayProvider> outputImage,
                       std::vector<Eigen::Vector4f> labelPositions,
                       int labelCount)
  : distancesImage(distancesImage), outputImage(outputImage),
    labelCount(labelCount), seedBuffer(labelCount)
{
  imageSize = outputImage->getWidth();

  for (size_t i = 0; i < seedBuffer.size(); ++i)
  {
    auto labelPosition = labelPositions[i];
    seedBuffer[i] = make_float4(labelPosition.x(), labelPosition.y(),
                                labelPosition.z(), labelPosition.w());
    int index = static_cast<int>(labelPosition.y()) +
                static_cast<int>(labelPosition.z()) * imageSize;
    pixelIndexToLabelId[index] = labelPosition.x();
  }

  pixelCount = imageSize * imageSize;

  distancesImage->map();
  auto resDesc = distancesImage->getResourceDesc();

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&distancesTexture, &resDesc, &texDesc, NULL);

  distancesImage->unmap();
}

void Apollonius::run()
{
  resize();
  outputImage->map();
  auto resDesc = outputImage->getResourceDesc();
  hipCreateSurfaceObject(&outputSurface, &resDesc);

  dimBlock = dim3(32, 32, 1);
  dimGrid = dim3(divUp(imageSize, dimBlock.x), divUp(imageSize, dimBlock.y), 1);

  runSeedKernel();
  runStepsKernels();
  runGatherKernel();

  outputImage->unmap();
}

thrust::device_vector<int> &Apollonius::getIds()
{
  return seedIds;
}

std::vector<int> Apollonius::calculateOrdering()
{
  extractUniqueBoundaryIndices();
  updateLabelSeeds();

  size_t iterationCount = 0;
  size_t labelCount = pixelIndexToLabelId.size();
  while (extractedIndices.size() < labelCount && iterationCount < labelCount)
  {
    run();
    extractUniqueBoundaryIndices();
    updateLabelSeeds();

    ++iterationCount;
  }

  return std::vector<int>(insertionOrder.begin(), insertionOrder.end());
}

void Apollonius::resize()
{
  if (computeVector.size() != static_cast<unsigned long>(pixelCount))
  {
    computeVector.resize(pixelCount, pixelCount);
  }

  if (seedIds.size() != labelCount || seedIndices.size() != labelCount)
  {
    seedIds.resize(labelCount, -1);
    seedIndices.resize(labelCount, -1);
  }
}

void Apollonius::runSeedKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  int *idPtr = thrust::raw_pointer_cast(seedIds.data());
  int *indicesPtr = thrust::raw_pointer_cast(seedIndices.data());
  float4 *seedBufferPtr = thrust::raw_pointer_cast(seedBuffer.data());

  seed<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount,
                              seedBufferPtr, computePtr, idPtr, indicesPtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runStepsKernels()
{
  apolloniusStep<<<dimGrid, dimBlock>>>
      (distancesTexture, thrust::raw_pointer_cast(computeVector.data()),
       1, imageSize, imageSize);

  for (int k = (imageSize / 2); k > 0; k /= 2)
  {
    apolloniusStep<<<dimGrid, dimBlock>>>
        (distancesTexture, thrust::raw_pointer_cast(computeVector.data()),
         k, imageSize, imageSize);
  }
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::runGatherKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  int *seedIdsPtr = thrust::raw_pointer_cast(seedIds.data());
  int *seedIndicesPtr = thrust::raw_pointer_cast(seedIndices.data());
  gather<<<dimGrid, dimBlock>>>(outputSurface, imageSize, labelCount,
      computePtr, seedIdsPtr, seedIndicesPtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void Apollonius::extractUniqueBoundaryIndices()
{
  const unsigned int borderSize = 4 * imageSize;

  if (orderedIndices.size() < borderSize)
  {
    orderedIndices.resize(borderSize);
  }

  dim3 dimBlock(32, 1, 1);
  dim3 dimGrid(divUp(imageSize, dimBlock.x), 1, 1);

  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  int *orderedIndicesPtr = thrust::raw_pointer_cast(orderedIndices.data());

  copyBorderIndex<<<dimGrid, dimBlock>>>(imageSize, computePtr,
      orderedIndicesPtr);

  HANDLE_ERROR(hipDeviceSynchronize());

  thrust::device_vector<int>::iterator it_found =
      thrust::unique(orderedIndices.begin(), orderedIndices.end());
  thrust::host_vector<int> uniqueIndices(orderedIndices.begin(), it_found);

  for (unsigned int i = 0; i < uniqueIndices.size(); i++)
  {
    const int uniqueIndex = uniqueIndices[i];
    if (uniqueIndex > 0)
    {
      if (extractedIndices.insert(uniqueIndex).second)
      {
        insertionOrder.push_front(pixelIndexToLabelId[uniqueIndex]);
      }
    }
  }
}

void Apollonius::updateLabelSeeds()
{
  thrust::host_vector<float4> labelsSeed = seedBuffer;
  for (size_t index = 0; index < labelsSeed.size(); ++index)
  {
    float4 seed = labelsSeed[index];
    const uint cindex =
        static_cast<int>(seed.y) + static_cast<int>(seed.z) * imageSize;

    if (extractedIndices.find(cindex) != extractedIndices.end())
    {
      if (seed.x > 0)
      {
        seed.x = -seed.x;
        labelsSeed[index] = seed;
      }
    }
  }

  seedBuffer = labelsSeed;
}

}  // namespace Placement

#include "hip/hip_runtime.h"
#include "./occlusion.h"
#include "../utils/cuda_helper.h"

__global__ void occupancyKernel(hipTextureObject_t positions,
                                hipSurfaceObject_t output, int width,
                                int height, int widthScale, int heightScale)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  float minTransparency = 0.0f;
  for (int i = 0; i < widthScale; ++i)
  {
    for (int j = 0; j < heightScale; ++j)
    {
      float4 color = tex2D<float4>(positions, x * widthScale + 0.5f + i,
                                      y * heightScale + 0.5f + j);
      if (color.w > minTransparency)
        minTransparency = color.w;
    }
  }

  surf2Dwrite(minTransparency, output, x * sizeof(float), y);
}

namespace Placement
{

Occlusion::Occlusion(std::vector<std::shared_ptr<CudaArrayProvider>> colorProviders,
                     std::shared_ptr<CudaArrayProvider> outputProvider)
  : colorProviders(colorProviders), outputProvider(outputProvider)
{
}

Occlusion::~Occlusion()
{
  if (positions)
    hipDestroyTextureObject(positions);
  if (output)
    hipDestroySurfaceObject(output);
}

void Occlusion::runKernel()
{
  if (!positions)
    createSurfaceObjects();

  float outputWidth = outputProvider->getWidth();
  float outputHeight = outputProvider->getHeight();

  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid(divUp(outputWidth, dimBlock.x), divUp(outputHeight, dimBlock.y),
               1);

  int widthScale = colorProviders[0]->getWidth() / outputWidth;
  int heightScale = colorProviders[0]->getHeight() / outputHeight;

  occupancyKernel<<<dimGrid, dimBlock>>>(positions, output,
      outputWidth, outputHeight, widthScale, heightScale);

  HANDLE_ERROR(hipDeviceSynchronize());
}

void Occlusion::createSurfaceObjects()
{
  colorProviders[0]->map();
  outputProvider->map();

  auto resDesc = colorProviders[0]->getResourceDesc();
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&positions, &resDesc, &texDesc, NULL);

  auto outputResDesc = outputProvider->getResourceDesc();
  hipCreateSurfaceObject(&output, &outputResDesc);

  colorProviders[0]->unmap();
  outputProvider->unmap();
}

}  // namespace Placement

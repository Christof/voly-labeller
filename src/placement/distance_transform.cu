#include "hip/hip_runtime.h"
#if _WIN32
#pragma warning(disable : 4244 4267)
#endif

#include "./distance_transform.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "../utils/cuda_helper.h"

/**
 * \brief Initializes the distance transform
 *
 * The value from the inputTexture is read. If it is equal to 0 the data value
 * is set to the index. Otherwise it is set to the given outlier value.
 */
__global__ void initializeForDistanceTransform(hipTextureObject_t input,
                                               int width, int height,
                                               float xscale, float yscale,
                                               int outlierValue, int *data)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;

  float pixelValue = tex2D<float>(input, x * xscale + 0.5f, y * yscale + 0.5f);

  data[index] = pixelValue == 0.0f ? index : outlierValue;
}

__global__ void distanceTransformStep(int *data, unsigned int step, int width,
                                      int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;

  int currentNearest = data[index];
  int currentY = currentNearest / width;
  int currentX = currentNearest - currentY * width;
  int currentDistance =
      (x - currentX) * (x - currentX) + (y - currentY) * (y - currentY);

  #pragma unroll
  for (int i = -1; i <= 1; i++)
  {
    int u = x + i * step;
    if (u < 0 || u >= width)
      continue;

    #pragma unroll
    for (int j = -1; j <= 1; j += 2 - i * i)
    {
      int v = y + j * step;
      if (v < 0 || v >= height)
        continue;

      int newIndex = v * width + u;
      int newNearest = data[newIndex];
      int newY = newNearest / width;
      int newX = newNearest - newY * width;
      int newDistance = (x - newX) * (x - newX) + (y - newY) * (y - newY);

      if (newDistance < currentDistance)
      {
        currentDistance = newDistance;
        currentNearest = newNearest;
      }
    }
  }

  data[index] = currentNearest;
}

__global__ void distanceTransformFinish(hipSurfaceObject_t output, int width,
                                        int height, int *data, float *result)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;
  int voronoival = data[index];

  int ty = voronoival / height;
  int tx = voronoival - ty * width;

  float sqdist = ((tx - x) * (tx - x) + (ty - y) * (ty - y));
  float resultValue = sqdist / (width * width + height * height);

  result[index] = resultValue;

  surf2Dwrite(resultValue, output, x * sizeof(float), y);
}

namespace Placement
{

DistanceTransform::DistanceTransform(
    std::shared_ptr<CudaArrayProvider> inputImage,
    std::shared_ptr<CudaArrayProvider> outputImage)
  : inputImage(inputImage), outputImage(outputImage)

{
  prepareInputTexture();
  prepareOutputSurface();
}

DistanceTransform::~DistanceTransform()
{
  if (inputTexture)
    hipDestroyTextureObject(inputTexture);
  if (outputSurface)
    hipDestroySurfaceObject(outputSurface);
}

void DistanceTransform::resize()
{
  pixelCount = outputImage->getWidth() * outputImage->getHeight();
  if (computeVector.size() != static_cast<unsigned long>(pixelCount))
  {
    computeVector.resize(pixelCount, pixelCount);
    resultVector.resize(pixelCount, pixelCount);
  }
}

void DistanceTransform::run()
{
  resize();

  dimBlock = dim3(32, 32, 1);
  dimGrid = dim3(divUp(outputImage->getWidth(), dimBlock.x),
                 divUp(outputImage->getHeight(), dimBlock.y), 1);

  runInitializeKernel();
  runStepsKernels();
  runFinishKernel();
}

thrust::device_vector<float> &DistanceTransform::getResults()
{
  return resultVector;
}

void DistanceTransform::prepareInputTexture()
{
  inputImage->map();
  auto resDesc = inputImage->getResourceDesc();

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&inputTexture, &resDesc, &texDesc, NULL);

  inputImage->unmap();
}

void DistanceTransform::prepareOutputSurface()
{
  outputImage->map();
  auto resDesc = outputImage->getResourceDesc();

  hipCreateSurfaceObject(&outputSurface, &resDesc);

  outputImage->unmap();
}

void DistanceTransform::runInitializeKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());

  // read depth buffer and initialize distance transform computation
  float xScale =
      static_cast<float>(inputImage->getWidth()) / outputImage->getWidth();
  float yScale =
      static_cast<float>(inputImage->getHeight()) / outputImage->getHeight();
  int outlierValue =
      (outputImage->getWidth() * 2) * (outputImage->getHeight() * 2) - 1;

  initializeForDistanceTransform<<<dimGrid, dimBlock>>>(inputTexture,
      outputImage->getWidth(), outputImage->getHeight(), xScale, yScale,
      outlierValue, computePtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void DistanceTransform::runStepsKernels()
{
  dimGrid = dim3(divUp(outputImage->getWidth(), dimBlock.x),
                 divUp(outputImage->getHeight(), dimBlock.y), 1);

  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  distanceTransformStep<<<dimGrid, dimBlock>>>(
      computePtr, 1, outputImage->getWidth(), outputImage->getHeight());

  for (int k = (outputImage->getWidth() / 2); k > 0; k /= 2)
  {
    distanceTransformStep<<<dimGrid, dimBlock>>>(
        computePtr, k, outputImage->getWidth(), outputImage->getHeight());
  }

  HANDLE_ERROR(hipDeviceSynchronize());
}

void DistanceTransform::runFinishKernel()
{
  // kernel which maps color to distance transform result
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  float *resultPtr = thrust::raw_pointer_cast(resultVector.data());

  distanceTransformFinish<<<dimGrid, dimBlock>>>(outputSurface,
      outputImage->getWidth(), outputImage->getHeight(), computePtr, resultPtr);

  HANDLE_ERROR(hipDeviceSynchronize());
}

}  // namespace Placement

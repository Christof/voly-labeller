#include "hip/hip_runtime.h"
#include "./distance_transform.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "../utils/cuda_helper.h"

surface<void, cudaSurfaceType2D> outputSurface;
texture<float, 2, hipReadModeElementType> inputTexture;

/**
 * \brief Initializes the distance transform
 *
 * The value from the inputTexture is read. If it is larger than or equel to
 * 0.99 the data value is set to the index. Otherwise it is set to the given
 * outlier value.
 */
__global__ void initializeForDistanceTransform(int width, int height,
    float xscale, float yscale, int outlierValue, int *data)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;

  float pixelValue = tex2D(inputTexture, x * xscale + 0.5f, y * yscale + 0.5f);

  data[index] = pixelValue >= 0.99f ? index : outlierValue;
}

__global__ void distanceTransformStep(int *data, unsigned int step, int width, int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;

  int currentNearest = data[index];
  int currentY = currentNearest / width;
  int currentX = currentNearest - currentY * width;
  int currentDistance =
      (x - currentX) * (x - currentX) + (y - currentY) * (y - currentY);

  #pragma unroll
  for (int i = -1; i <= 1; i++)
  {
    int u = x + i * step;
    if (u < 0 || u >= width)
      continue;

    #pragma unroll
    for (int j = -1; j <= 1; j += 2 - i * i)
    {
      int v = y + j * step;
      if (v < 0 || v >= height)
        continue;

      int newIndex = v * width + u;
      int newNearest = data[newIndex];
      int newY = newNearest / width;
      int newX = newNearest - newY * width;
      int newDistance = (x - newX) * (x - newX) + (y - newY) * (y - newY);

      if (newDistance < currentDistance)
      {
        currentDistance = newDistance;
        currentNearest = newNearest;
      }
    }
  }

  data[index] = currentNearest;
}

__global__ void distanceTransformFinish(int width, int height, int *data,
                                                float *result)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;
  int voronoival = data[index];

  int ty = voronoival / height;
  int tx = voronoival - ty * width;

  float sqdist = ((tx - x) * (tx - x) + (ty - y) * (ty - y));
  float distf = sqrtf(sqdist);

  result[index] = distf;

  // write to texture for debugging
  float4 color =
      make_float4(16.0f * distf / width, 16.0f * distf / width,
                  16.0f * distf / width, 1.0f);
  surf2Dwrite<float4>(color, outputSurface, x * sizeof(float4), y);
}

/*
void
cudaJFADistanceTransformThrust(std::shared_ptr<CudaArrayProvider> inputImage,
                               std::shared_ptr<CudaArrayProvider> outputImage,
                               int image_size, int screen_size_x,
                               int screen_size_y,
                               thrust::device_vector<int> &compute_vector,
                               thrust::device_vector<float> &result_vector)
{
  inputImage->map();
  outputImage->map();

  cudaJFADistanceTransformThrust(inputImage->getArray(), inputImage->getChannelDesc(),
                                 outputImage->getArray(), image_size, screen_size_x,
                                 screen_size_y, compute_vector,
                                 result_vector);

  outputImage->unmap();
  inputImage->unmap();
}
*/

DistanceTransform::DistanceTransform(
    std::shared_ptr<CudaArrayProvider> inputImage,
    std::shared_ptr<CudaArrayProvider> outputImage)
  : inputImage(inputImage), outputImage(outputImage)

{
}

void DistanceTransform::resize()
{
  pixelCount = outputImage->getWidth() * outputImage->getHeight();
  if (computeVector.size() != static_cast<unsigned long>(pixelCount))
  {
    computeVector.resize(pixelCount, pixelCount);
    resultVector.resize(pixelCount, pixelCount);
  }
}

void DistanceTransform::run()
{
  resize();
  inputImage->map();

  /*
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = inputImage->getArray();

  // Specify texture object parameters struct hipTextureDesc
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&inputTexture, &resDesc, &texDesc, NULL);
  */

  outputImage->map();
  HANDLE_ERROR(hipBindTextureToArray(inputTexture, inputImage->getArray(),
                                      inputImage->getChannelDesc()));
  HANDLE_ERROR(cudaBindSurfaceToArray(outputSurface, outputImage->getArray(),
                                      outputImage->getChannelDesc()));
  dimBlock = dim3(32, 32, 1);
  dimGrid = dim3(divUp(inputImage->getWidth(), dimBlock.x),
               divUp(inputImage->getHeight(), dimBlock.y), 1);

  runInitializeKernel();
  runStepsKernels();
  runFinishKernel();

  // hipDestroyTextureObject(inputTexture);
  hipUnbindTexture(&inputTexture);
  inputImage->unmap();
}

thrust::device_vector<float> &DistanceTransform::getResults()
{
  return resultVector;
}

void DistanceTransform::runInitializeKernel()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());

  // read depth buffer and initialize distance transform computation

  float xScale =
      static_cast<float>(inputImage->getWidth()) / outputImage->getWidth();
  float yScale =
      static_cast<float>(inputImage->getHeight()) / outputImage->getHeight();
  int outlierValue =
      (outputImage->getWidth() * 2) * (outputImage->getHeight() * 2) - 1;

  initializeForDistanceTransform<<<dimGrid, dimBlock>>>(//inputTexture,
      outputImage->getWidth(), outputImage->getHeight(), xScale, yScale, 
      outlierValue, computePtr);
  HANDLE_ERROR(hipDeviceSynchronize());
}

void DistanceTransform::runStepsKernels()
{
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  distanceTransformStep<<<dimGrid, dimBlock>>>(
      computePtr, 1, outputImage->getWidth(), outputImage->getHeight());

  for (int k = (outputImage->getWidth() / 2); k > 0; k /= 2)
  {
    distanceTransformStep<<<dimGrid, dimBlock>>>(
        computePtr, k, outputImage->getWidth(), outputImage->getHeight());
  }

  HANDLE_ERROR(hipDeviceSynchronize());
}

void DistanceTransform::runFinishKernel()
{
  // kernel which maps color to distance transform result
  int *computePtr = thrust::raw_pointer_cast(computeVector.data());
  float *resultPtr = thrust::raw_pointer_cast(resultVector.data());

  distanceTransformFinish<<<dimGrid, dimBlock>>>(
      outputImage->getWidth(), outputImage->getHeight(), computePtr, resultPtr);

  HANDLE_ERROR(hipDeviceSynchronize());
}

void cudaJFADistanceTransformThrust(
    hipArray_t inputImageArray, hipChannelFormatDesc inputImageDesc,
    hipArray_t outputImageArray, int image_size, int screen_size_x,
    int screen_size_y, thrust::device_vector<int> &compute_vector,
    thrust::device_vector<float> &result_vector)
{
  if (compute_vector.size() !=
      static_cast<unsigned long>(image_size * image_size))
  {
    compute_vector.resize(image_size * image_size, image_size * image_size);
    result_vector.resize(image_size * image_size, image_size * image_size);
  }

  int *compute_index_ptr = thrust::raw_pointer_cast(compute_vector.data());
  float *result_value_ptr = thrust::raw_pointer_cast(result_vector.data());

  dim3 dimBlock(32, 1, 1);
  dim3 dimGrid(divUp(image_size, dimBlock.x), divUp(image_size, dimBlock.y), 1);
  // read depth buffer and initialize distance transform computation
  inputTexture.normalized = 0;
  inputTexture.filterMode = hipFilterModeLinear;
  inputTexture.addressMode[0] = hipAddressModeWrap;
  inputTexture.addressMode[1] = hipAddressModeWrap;

  hipBindTextureToArray(&inputTexture, inputImageArray, &inputImageDesc);
  float xScale = static_cast<float>(screen_size_x) / image_size;
  float yScale = static_cast<float>(screen_size_y) / image_size;
  int outlierValue = (image_size * 2) * (image_size * 2) - 1;

  initializeForDistanceTransform<<<dimGrid, dimBlock>>>(image_size,
      image_size, xScale, yScale, outlierValue, compute_index_ptr);
  hipDeviceSynchronize();

  hipUnbindTexture(&inputTexture);

  // voronoi diagram computation in thrust
  distanceTransformStep<<<dimGrid, dimBlock>>>(
      thrust::raw_pointer_cast(compute_vector.data()), 1, image_size,
      image_size);

  for (int k = (image_size / 2); k > 0; k /= 2)
  {
    distanceTransformStep<<<dimGrid, dimBlock>>>(
        thrust::raw_pointer_cast(compute_vector.data()), k, image_size,
        image_size);
  }

  hipDeviceSynchronize();

  hipChannelFormatDesc outputChannelDesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  HANDLE_ERROR(cudaBindSurfaceToArray(outputSurface, outputImageArray,
                                      outputChannelDesc));

  // kernel which maps color to distance transform result
  compute_index_ptr = thrust::raw_pointer_cast(compute_vector.data());
  distanceTransformFinish<<<dimGrid, dimBlock>>>(
      image_size, image_size, compute_index_ptr, result_value_ptr);

  hipDeviceSynchronize();
}

